#include "hip/hip_runtime.h"
#include "../include/effect.cuh"

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace mop;

namespace mop_cuda {

	__host__ __device__ __forceinline__ int divUp(int total, int grain) {
		return (total + grain - 1) / grain;
	}

	__device__ void random(int* data, int max, int seed) {

		hiprandState_t state;
		hiprand_init(seed, 0, 0, &state);
		*data = hiprand(&state) % max;

	}

	__global__ void NoiseDevice(uchar* src, uchar* dst, int w, int h, int c) {

		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;



	}

	DLL_EXPORT void Noise(matrix* src, matrix* dst, int percent, int seed) {


		int w = src->width(),
			h = src->height(),
			c = src->channel();

		uchar *input, *output;
		int size = sizeof(uchar) * w * h * c;

		hipMalloc((void**)&input, size);
		hipMalloc((void**)&output, size);

		hipMemcpy(input, src->data, size, hipMemcpyHostToDevice);

		*dst = matrix(w, h, c);

		const dim3 block(w / 125, h / 125);
		const dim3 grid(divUp(w, block.x), divUp(h, block.y));

		NoiseDevice << <grid, block >> > (input, output, w, h, c);

		hipMemcpy(
			(void*)dst->data,
			(void*)output,
			size,
			hipMemcpyDeviceToHost
		);

		hipFree(input);
		hipFree(output);

	}

}
