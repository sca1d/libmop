#include "hip/hip_runtime.h"
/*
#include <hip/hip_runtime.h>
#include "hip/hip_vector_types.h"
#include "hip/hip_runtime.h"
#include ""
*/
#include <stdio.h>
#include <math.h>

#include <iostream>

#include "cuda.cuh"
#include "hip/hip_math_constants.h"

__global__ void gpu_func(float* dx, float* dy) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	dy[i] = sin(dx[i]) * sin(dx[i]) + cos(dx[i]) * cos(dx[i]);

}

__host__ void test(void) {

    int N = 1000000;
    float* host_x, * host_y, * dev_x, * dev_y;
    host_x = (float*)malloc(N * sizeof(float));
    host_y = (float*)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++) {
        host_x[i] = rand();
    }

    hipMalloc(&dev_x, N * sizeof(float));
    hipMalloc(&dev_y, N * sizeof(float));

    hipMemcpy(dev_x, host_x, N * sizeof(float), hipMemcpyHostToDevice);

    gpu_func << <(N + 255) / 256, 256 >> > (dev_x, dev_y);

    hipMemcpy(host_y, dev_y, N * sizeof(float), hipMemcpyDeviceToHost);

    float sum = 0.0f;
    for (int j = 0; j < N; j++) {
        sum += host_y[j];
    }
    std::cout << sum << std::endl;

}
