#include "hip/hip_runtime.h"
#include "../include/gpu_calc.cuh"

DLL_EXPORT __host__ void initialize_uchar(unsigned char** data, int w, int h, int c) {

	unsigned char* dst;

	int size = sizeof(unsigned char) * w * h * c;

	hipMalloc((void**)&dst, size);

	data_init_gpu_uchar << <w * h, c >> > (dst);

	hipMemcpy(*data, dst, size, hipMemcpyDeviceToHost);

}
DLL_EXPORT __host__ void initialize_ushort(unsigned short** data, int w, int h, int c) {

	unsigned short* dst;

	int size = sizeof(unsigned short) * w * h * c;

	hipMalloc((void**)&dst, size);

	data_init_gpu_ushort << <w * h, c >> > (dst);

	hipMemcpy(*data, dst, size, hipMemcpyDeviceToHost);

}
DLL_EXPORT __host__ void initialize_float(float** data, int w, int h, int c) {

	float* dst;

	int size = sizeof(float) * w * h * c;

	hipMalloc((void**)&dst, size);

	data_init_gpu_float << <w * h, c >> > (dst);

	hipMemcpy(*data, dst, size, hipMemcpyDeviceToHost);

}

__global__ void data_init_gpu_uchar(unsigned char* dst) {

	int ID = blockIdx.x * blockDim.x + threadIdx.x;
	dst[ID] = 100;

}
__global__ void data_init_gpu_ushort(unsigned short* dst) {

	int ID = blockIdx.x * blockDim.x + threadIdx.x;
	dst[ID] = 0x8000;

}
__global__ void data_init_gpu_float(float* dst) {

	int ID = blockIdx.x * blockDim.x + threadIdx.x;
	dst[ID] = 200.0f;

}
