#include "hip/hip_runtime.h"
#include "../include/gpu_calc.cuh"

DLL_EXPORT __host__ void initialize(unsigned char** data, int w, int h, int c) {

	unsigned char* dst;

	int size = w * h * c;

	hipMalloc((void**)&dst, sizeof(unsigned char) * size);

	data_init_gpu << <size, c >> > (dst);

	hipMemcpy(data, dst, sizeof(unsigned char) * size, hipMemcpyDeviceToHost);

}

__global__ void data_init_gpu(unsigned char* dst) {

	int ID = blockIdx.x * blockDim.x + threadIdx.x;
	dst[ID] = 0x00;

}